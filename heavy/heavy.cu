#include "hip/hip_runtime.h"
#include <string.h>
#include <openssl/sha.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <map>

#ifndef _WIN32
#include <unistd.h>
#endif

// include thrust
#include <thrust/version.h>
#include <thrust/remove.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/constant_iterator.h>

#include "miner.h"

extern "C" char *device_name[8];

// nonce-array f�r die threads
uint32_t *d_nonceVector[8];

/* Combines top 64-bits from each hash into a single hash */
static void combine_hashes(uint32_t *out, const uint32_t *hash1, const uint32_t *hash2, const uint32_t *hash3, const uint32_t *hash4)
{
    const uint32_t *hash[4] = { hash1, hash2, hash3, hash4 };
    int bits;
    unsigned int i;
    uint32_t mask;
    unsigned int k;

    /* Transpose first 64 bits of each hash into out */
    memset(out, 0, 32);
    bits = 0;
    for (i = 7; i >= 6; i--) {
        for (mask = 0x80000000; mask; mask >>= 1) {
            for (k = 0; k < 4; k++) {
                out[(255 - bits)/32] <<= 1;
                if ((hash[k][i] & mask) != 0)
                    out[(255 - bits)/32] |= 1;
                bits++;
            }
        }
    }
}

#ifdef _MSC_VER
#include <intrin.h>
static uint32_t __inline bitsset( uint32_t x )
{
   DWORD r = 0;
   _BitScanReverse(&r, x);
   return r;
}
#else
static uint32_t bitsset( uint32_t x )
{
    return 31-__builtin_clz(x);
}
#endif

// Finde das high bit in einem Multiword-Integer.
static int findhighbit(const uint32_t *ptarget, int words)
{
    int i;
    int highbit = 0;
    for (i=words-1; i >= 0; --i)
    {
        if (ptarget[i] != 0) {
            highbit = i*32 + bitsset(ptarget[i])+1;
                break;
        }
    }
    return highbit;
}

// Generiere ein Multiword-Integer das die Zahl
// (2 << highbit) - 1 repr�sentiert.
static void genmask(uint32_t *ptarget, int words, int highbit)
{
    int i;
    for (i=words-1; i >= 0; --i)
    {
        if ((i+1)*32 <= highbit)
            ptarget[i] = 0xffffffff;
        else if (i*32 > highbit)
            ptarget[i] = 0x00000000;
        else
            ptarget[i] = (1 << (highbit-i*32)) - 1;
    }
}

struct check_nonce_for_remove
{    
    check_nonce_for_remove(uint64_t target, uint32_t *hashes, uint32_t hashlen, uint32_t startNonce) :
        m_target(target),
        m_hashes(hashes),
        m_hashlen(hashlen),
        m_startNonce(startNonce) { }

    __device__
    bool operator()(const uint32_t x)
    {
        // Position im Hash Buffer
        uint32_t hashIndex = x - m_startNonce;
        // Wert des Hashes (als uint64_t) auslesen.
        // Steht im 6. und 7. Wort des Hashes (jeder dieser Hashes hat 512 Bits)
        uint64_t hashValue = *((uint64_t*)(&m_hashes[m_hashlen*hashIndex + 6]));
        // gegen das Target pr�fen. Es d�rfen nur Bits aus dem Target gesetzt sein.
        return (hashValue & m_target) != hashValue;
    }

    uint64_t  m_target;
    uint32_t *m_hashes;
    uint32_t  m_hashlen;
    uint32_t  m_startNonce;
};

// Zahl der CUDA Devices im System bestimmen
extern "C" int cuda_num_devices()
{
    int version;
    hipError_t err = hipDriverGetVersion(&version);
    if (err != hipSuccess)
    {
     //   applog(LOG_ERR, "Unable to query CUDA driver version! Is an nVidia driver installed?");
        exit(1);
    }

    int maj = version / 1000, min = version % 100; // same as in deviceQuery sample
    if (maj < 5 || (maj == 5 && min < 5))
    {
    //    applog(LOG_ERR, "Driver does not support CUDA %d.%d API! Update your nVidia driver!", 5, 5);
        exit(1);
    }

    int GPU_N;
    err = hipGetDeviceCount(&GPU_N);
    if (err != hipSuccess)
    {
     //   applog(LOG_ERR, "Unable to query number of CUDA devices! Is an nVidia driver installed?");
        exit(1);
    }
    return GPU_N;
}

// Ger�tenamen holen
extern char *device_name[8];
extern int device_map[8];
int device_major[8];
int device_minor[8];

extern "C" void cuda_devicenames()
{
    hipError_t err;
    int GPU_N;
    err = hipGetDeviceCount(&GPU_N);
    if (err != hipSuccess)
    {
     //   applog(LOG_ERR, "Unable to query number of CUDA devices! Is an nVidia driver installed?");
        exit(1);
    }

    for (int i=0; i < GPU_N; i++)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, device_map[i]);

        device_name[i] = strdup(props.name);
		device_major[i] = props.major; 
		device_minor[i] = props.minor;
    }
}

static bool substringsearch(const char *haystack, const char *needle, int &match)
{
    int hlen = strlen(haystack);
    int nlen = strlen(needle);
    for (int i=0; i < hlen; ++i)
    {
        if (haystack[i] == ' ') continue;
        int j=0, x = 0;
        while(j < nlen)
        {
            if (haystack[i+x] == ' ') {++x; continue;}
            if (needle[j] == ' ') {++j; continue;}
            if (needle[j] == '#') return ++match == needle[j+1]-'0';
            if (tolower(haystack[i+x]) != tolower(needle[j])) break;
            ++j; ++x;
        }
        if (j == nlen) return true;
    }
    return false;
}

// CUDA Ger�t nach Namen finden (gibt Ger�te-Index zur�ck oder -1)
extern "C" int cuda_finddevice(char *name)
{
    int num = cuda_num_devices();
    int match = 0;
    for (int i=0; i < num; ++i)
    {
        hipDeviceProp_t props;
        if (hipGetDeviceProperties(&props, i) == hipSuccess)
            if (substringsearch(props.name, name, match)) return i;
    }
    return -1;
}

// Zeitsynchronisations-Routine von cudaminer mit CPU sleep
typedef struct { double value[8]; } tsumarray;
hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id)
{
    hipError_t result = hipSuccess;
    if (situation >= 0)
    {   
        static std::map<int, tsumarray> tsum;

        double a = 0.95, b = 0.05;
        if (tsum.find(situation) == tsum.end()) { a = 0.5; b = 0.5; } // faster initial convergence

        double tsync = 0.0;
        double tsleep = 0.95 * tsum[situation].value[thr_id];
        if (hipStreamQuery(stream) == hipErrorNotReady)
        {
            usleep((useconds_t)(1e6*tsleep));
            struct timeval tv_start, tv_end;
            gettimeofday(&tv_start, NULL);
            result = hipStreamSynchronize(stream);
            gettimeofday(&tv_end, NULL);
            tsync = 1e-6 * (tv_end.tv_usec-tv_start.tv_usec) + (tv_end.tv_sec-tv_start.tv_sec);
        }
        if (tsync >= 0) tsum[situation].value[thr_id] = a * tsum[situation].value[thr_id] + b * (tsleep+tsync);
    }
    else
        result = hipStreamSynchronize(stream);
    return result;
}


extern bool opt_benchmark;

